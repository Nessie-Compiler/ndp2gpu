#include "hip/hip_runtime.h"
#include "config.h"
#include "vcode.h"
#include <cvl.h>
#include "y.tab.h"
#include <cutil_inline.h>
#include "defins.cuh"

MAXALIGN *ComputeMemory = NULL;

extern "C" void init (MAXALIGN *mem) {
  ComputeMemory = mem;
}

__global__ void fused0Kernel(MAXALIGN *data, int dst, int s0, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  int *pDst = (int*)(&data[dst]);
  int *pSrc0 = (int*)(&data[s0]);
  
  if (address < len) {
    pDst[address] = (cvlrand((100)));
  }
}

__global__ void fused1Kernel(MAXALIGN *data, int dst, int s0, int s1, int s2, int len, int scratch) {
  int address = blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
  int *pDst = (int*)(&data[dst]);
  int *pSrc0 = (int*)(&data[s0]);
  int *pSrc1 = (int*)(&data[s1]);
  int *pSrc2 = (int*)(&data[s2]);
  
  if (address < len) {
    pDst[address] = (plus((times(pSrc0[address], pSrc1[address])), pSrc2[address]));
  }
}

void fused0(vec_p d, vec_p s0, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused0Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, len, scratch);
  cutilCheckMsg("fused0 execution failed\n");
}

void fused1(vec_p d, vec_p s0, vec_p s1, vec_p s2, int len, vec_p scratch) {
  if (len==0) {return;}
  SYNC();
  DEF_BLOCKS_PER_GRID(len);
  fused1Kernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(ComputeMemory, d, s0, s1, s2, len, scratch);
  cutilCheckMsg("fused1 execution failed\n");
}

make_no_scratch(fused0)
make_no_scratch(fused1)
make_inplace(fused0, INPLACE_NONE)
make_inplace(fused1, INPLACE_NONE)
vopdes_t vops[] = {
  {FUSED, "fused0", 0, 1,
  {Int,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Int,},
  {AGREE1,},
  {1,},
  Elwise1},
  {FUSED, "fused1", 0, 1,
  {Int,Int,Int,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {NONE,NONE,NONE,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal,Illegal},
  {Int,},
  {AGREE1,},
  {1,},
  Elwise3},
  };

cvl_triple_t cvl_funs[] = {
  { { (void (*)())fused0, (int (*)())fused0_scratch, (unsigned (*)())fused0_inplace },},
  { { (void (*)())fused1, (int (*)())fused1_scratch, (unsigned (*)())fused1_inplace },},
  };
